//
//  CrossCorrelation.cu
//  CrossCorrelation
//
//  Created by Vivek Sridhar on 29/06/17.
//  Copyright © 2017 Vivek Sridhar. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <dirent.h>

template <typename T>
std::string to_string(const T& value) {
    std::stringstream ss;
    ss << value;
    return ss.str();
}

long factorial(long val)
{
std::cout << val << "\n";
    long result = 1;
    for (long i = 1; i <= val; ++i)
    {
        result *= i;
    }

    return result;
}

long combination(long n, long r)
{
    return (factorial(n)) / ((factorial(n - r)) * factorial(r));
}

__global__ void kernel(float *x1, float *y1, float *x2, float *y2, float *res, int tau, int na_frames, long nElements)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (tau < 0)
    {
        if (index >= -tau+na_frames)
        {
            res[index] = x1[index] * x2[index + tau] + y1[index] * y2[index + tau];
        }
	   else res[index] = 0.0;
    }
    else
    {
        if (index < nElements - tau)
        {
            res[index] = x1[index] * x2[index + tau] + y1[index] * y2[index + tau];
        }
	   else res[index] = 0.0;
    }
}

// total measurement points in the time series is defined by nElements
#define M 1024      // number of threads per block
#define fps 10      // frames per second of input video (used to determine tau)
#define time 5      // time in seconds within which time delayed cross correlation is calculated (tau ranges from -time*fps to time*fps)
#define n_inds 10

int na_frames = 0;  // number of frames in the start with nas
int scale = 1;    // time window for analysis in seconds; varying this allows us to examine dynamics of leadership across varying timescales; setting scale larger than the entire time series or -1 gives aggregated statistics across the entire duration (otherwise, timescale of analysis is scale*fps)

//const int pairs = combination(n_inds, 2);
const bool aggregate = false;	// this boolean decides whether you output a dynamic time variable leadership network or a static time aggregated network; scale is set to -1 if aggregate is true

std::ofstream outputFile1;

int main () 
{
    DIR *dir;
    FILE *pFile_x1; FILE *pFile_y1; FILE *pFile_x2; FILE *pFile_y2;
    long lSize;
    long nElements;
    struct dirent *file;
    
    float *d_x1, *d_y1, *d_x2, *d_y2, *d_res;
    float *x1, *y1, *x2, *y2, *res;

    size_t result_x1, result_y1, result_x2, result_y2;

    if (aggregate) scale = -1;
    
    std::vector<std::string> files;
    std::string directory = "/home/user/Documents/Vivek/cuda/DirectionalCorrelation/Data/Input/pigeons/10_birds/ffA3/cross_correlation/";
    dir = opendir(directory.c_str());

    int idx = 0;
    while ((file = readdir(dir)) != NULL)
    {
        if (file->d_name[0] == 'd')
        {
            files.push_back(file->d_name);
            ++idx;
        }
    }
    std::sort(files.begin(), files.begin()+2*n_inds);
    closedir(dir);

    // Open output file
    std::string filename_cc;
    if (scale != -1) filename_cc = "cross_correlation_01.csv";
    else filename_cc = "avgd_cross_correlation.csv";
    outputFile1.open(filename_cc.c_str());

    // Output file headers
    if (aggregate || scale == -1) outputFile1 << "id1"<< ", " << "id2"  << ", " << "tau" << ", " << "cc" << "\n";
    else outputFile1 << "time" << ", " << "id1" << ", " << "id2" << ", " << "tau" << ", " << "cc" << "\n";

    //files = {"dir_x00", "dir_x01", "dir_y00", "dir_y01"}

    for (int a = 0; a < n_inds; ++a)
    {
		for (int b = 0; b < n_inds; ++b)
		{
			if (b != a)
			{
				pFile_x1 = fopen ((directory + files[a]).c_str(), "rb");
	        	pFile_y1 = fopen ((directory + files[a+n_inds]).c_str(), "rb");
	        	pFile_x2 = fopen ((directory + files[b]).c_str(), "rb");
	        	pFile_y2 = fopen ((directory + files[b+n_inds]).c_str(), "rb");
	        	if (pFile_x1==NULL || pFile_y1==NULL || pFile_x2==NULL || pFile_y2==NULL) { fputs ("File error",stderr); exit (1); }
	        
        		// obtain file size
        		fseek (pFile_x1 , 0 , SEEK_END);
        		lSize = ftell (pFile_x1);
        		rewind (pFile_x1);
        
       			nElements = lSize / sizeof(float);

        		// allocate memory to contain the whole file
        		// device memory
        		hipMalloc((void **) &d_x1, lSize);
        		hipMalloc((void **) &d_y1, lSize);
        		hipMalloc((void **) &d_x2, lSize);
        		hipMalloc((void **) &d_y2, lSize);
        		hipMalloc((void **) &d_res, lSize);

        		// host memory
        		x1 = (float*) malloc(lSize);
        		y1 = (float*) malloc(lSize);
        		x2 = (float*) malloc(lSize);
        		y2 = (float*) malloc(lSize);
        		res = (float*) malloc(lSize);
        		if (x1 == NULL || y1==NULL || x2==NULL || y2==NULL || res==NULL) { fputs ("Memory error",stderr); exit (2); }

        		// copy the file into the respective float pointers
        		result_x1 = fread (x1, sizeof(float), nElements, pFile_x1);
        		result_y1 = fread (y1, sizeof(float), nElements, pFile_y1);
        		result_x2 = fread (x2, sizeof(float), nElements, pFile_x2);
        		result_y2 = fread (y2, sizeof(float), nElements, pFile_y2);
        		if (result_x1 != nElements || result_y1 != nElements || result_x2 != nElements || result_y2 != nElements) { fputs ("Reading error",stderr); exit (3); }
        
        		// the whole files are now loaded in the memory x1, y1, x2 and y2 respectively
        
        		hipMemcpy(d_x1, x1, lSize, hipMemcpyHostToDevice);
        		hipMemcpy(d_y1, y1, lSize, hipMemcpyHostToDevice);
       			hipMemcpy(d_x2, x2, lSize, hipMemcpyHostToDevice);
        		hipMemcpy(d_y2, y2, lSize, hipMemcpyHostToDevice);
			
				if (scale*fps > nElements) scale = -1;

                int tau_max[nElements - scale*fps];
                float res_tmp[nElements - scale*fps];
                float res_max[nElements - scale*fps];
                std::fill_n(tau_max, nElements - scale*fps, 0);
                std::fill_n(res_tmp, nElements - scale*fps, 0.0);
                std::fill_n(res_max, nElements - scale*fps, -1.0);

				for (int tau = -time*fps; tau <= time*fps; ++tau) 
                {
                    kernel<<<(nElements + M - 1) / M, M>>>(d_x1, d_y1, d_x2, d_y2, d_res, tau, na_frames, nElements);
                    hipMemcpy(res, d_res, lSize, hipMemcpyDeviceToHost);

                    if (scale == -1)
                    {
                        float res_now = -1.0f;
                        for (int i = na_frames; i < nElements; ++i) 
			{
				if (res[i] != res[i]) std::cout << x1[i] << " " << y1[i] << " " << i << " " << tau << "\n";       // if nans
				res_now += res[i];
			}

                        outputFile1 << (to_string(files[a][5])).c_str() << (to_string(files[a][6])).c_str() << (to_string(files[a][7])).c_str()  << ", " << (to_string(files[b][5])).c_str() << (to_string(files[b][6])).c_str() << (to_string(files[b][7])).c_str() << ", " << tau << ", " << res_now / nElements << "\n";
                    }
                    else
                    {
			std::fill_n(res_tmp, nElements - scale*fps, 0.0);
                        for (int i = na_frames; i < nElements - scale*fps; ++i)
                        {
                            for (int j = i; j < i + scale*fps; ++j)
                            {
                                res_tmp[i] += res[j];
                                if (j == i + scale*fps - 1 && res_max[i] < res_tmp[i]) { res_max[i] = res_tmp[i]; tau_max[i] = tau; }
                            }
                        }
                    }
                }

                if (scale != -1) 
                {
                    for (int t = 0; t < nElements - scale*fps; ++t)
                    {
                        outputFile1 << t  + scale*fps/2 << ", " << (to_string(files[a][5])).c_str() << (to_string(files[a][6])).c_str() << (to_string(files[a][7])).c_str() << ", " << (to_string(files[b][5])).c_str() << (to_string(files[b][6])).c_str() << (to_string(files[b][7])).c_str() << ", " << tau_max[t] << ", " << res_max[t] / (scale*fps) << "\n";
                    }
                }

		fclose(pFile_x1);
		fclose(pFile_x2);
		fclose(pFile_y1);
		fclose(pFile_y2);

		hipFree(d_x1); hipFree(d_y1); hipFree(d_x2); hipFree(d_y2); hipFree(d_res);
		free(x1); free(y1); free(x2); free(y2);
			}
		}
    }
    
    // terminate
    fclose(pFile_x1); fclose(pFile_y1); fclose(pFile_x2); fclose(pFile_y2);
    
    return 0;
}
